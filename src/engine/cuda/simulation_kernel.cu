#include "hip/hip_runtime.h"
// Internal
#include "../engine.hpp"
#include "cuda_utils.hpp"
#include "collision_detection.cuh"
#include "physics_update.cuh"

// C++
#include <string>
#include <iostream>
#include <fstream>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <stdint.h>
#include <thread>

// CUDA
#include <hip/hip_vector_types.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <nvfunctional>

using namespace std::chrono_literals; // ns, us, ms, s, h, etc.
using std::chrono::system_clock;


namespace STARDUST {

	void DEMEngine::cleanBuffers() {
		CUDA_ERR_CHECK(hipDeviceSynchronize());

		CUDA_ERR_CHECK(hipMemset(
			d_particle_forces_ptr,
			0,
			m_num_particles * sizeof(float4))
		);

		CUDA_ERR_CHECK(hipMemset(
			d_rigid_body_forces_ptr,
			0,
			m_num_entities * sizeof(float4))
		);

		CUDA_ERR_CHECK(hipMemset(
			d_rigid_body_torques_ptr,
			0,
			m_num_entities * sizeof(float4))
		);
	}

	void DEMEngine::step(Scalar timestep) {

		// Steps
		// 1. Update sphere positions and velocities (rotation and translation) based on new particle position
		// 2. Compute relative positions of spheres to entity COM
		// 3. Compute collisions
		// 4. Compute particle force and torques from spheres
		// 5. Compute momentum (linear and angular) on particle
		// 6. Advect particle and compute quaternion

		float cell_dim = 0.2 * 2;

		int threads_per_block = 128;
		unsigned int particle_size = (m_num_particles - 1) / threads_per_block + 1;

		// PARTICLE UPDATES AND RELATIVE POSITIONS
		updateSphereData(
			m_num_particles,
			m_num_entities,
			d_particle_position_ptr,
			d_particle_velocity_ptr,
			d_particle_init_relative_position_ptr,
			d_particle_relative_position_ptr,
			d_particle_to_rigid_idx_ptr,
			d_rigid_body_position_ptr,
			d_rigid_body_velocity_ptr,
			d_rigid_body_quaternion_ptr,
			d_rigid_body_angular_velocity_ptr,
			particle_size
		);

		cleanBuffers();

		// COLLISION DETECTION AND RESPONSE //
		if (spatialHashCollision) {

			std::chrono::time_point<std::chrono::system_clock> start;
			std::chrono::duration<double> duration;

			double time;
			start = std::chrono::system_clock::now();

			SpatialPartition::constructCollisionList(
				m_num_particles,
				cell_dim,
				d_grid_ptr,
				d_sphere_ptr,
				d_particle_size_ptr,
				d_particle_position_ptr,
				threads_per_block,
				d_temp_ptr
			);

			SpatialPartition::sortCollisionList(
				d_grid_ptr,
				d_sphere_ptr,
				d_grid_temp_ptr,
				d_sphere_temp_ptr,
				d_radices_ptr,
				d_radix_sums_ptr,
				m_num_particles
			);


			SpatialPartition::tranverseAndResolveCollisionList(
				d_grid_ptr,
				d_sphere_ptr,
				d_particle_position_ptr,
				d_particle_velocity_ptr,
				d_particle_forces_ptr,
				d_particle_mass_ptr,
				d_particle_size_ptr,
				d_particle_to_rigid_idx_ptr,
				m_num_particles,
				d_temp_ptr,
				threads_per_block
			);

			duration = std::chrono::system_clock::now() - start;

			time = duration.count();

			std::cout << "Done Collision analysis completed in: " << time << "s on " << m_num_particles << " particles\n";

		}
		else if (LBVHCollision) {

		}

		/*float4 force0;
		float4 force1;
		CUDA_ERR_CHECK(hipMemcpy(&force0, d_particle_forces_ptr, sizeof(float4), hipMemcpyDeviceToHost));
		CUDA_ERR_CHECK(hipMemcpy(&force1, d_particle_forces_ptr + 1, sizeof(float4), hipMemcpyDeviceToHost));

		printf("Force on Particle 0: %.3f, %.3f, %.3f\n", force0.x, force0.y, force0.z);
		printf("Force on Particle 1: %.3f, %.3f, %.3f\n", force1.x, force1.y, force1.z);*/

		// PARTICLE FORCE COMPUTATION AND POSITION/ORIENTATION UPDATE

		computeForcesAndTorquesOnParticles(
			m_num_particles,
			m_num_entities,
			d_rigid_body_forces_ptr,
			d_rigid_body_torques_ptr,
			d_rigid_body_mass_ptr,
			d_entity_start_ptr,
			d_entity_length_ptr,
			d_particle_relative_position_ptr,
			d_particle_forces_ptr,
			d_particle_to_rigid_idx_ptr
		);

		advectParticles(
			m_num_particles,
			m_num_entities,
			timestep,
			d_rigid_body_position_ptr,
			d_rigid_body_velocity_ptr,
			d_rigid_body_angular_velocity_ptr,
			d_rigid_body_forces_ptr,
			d_rigid_body_torques_ptr,
			d_rigid_body_quaternion_ptr,
			d_rigid_body_mass_ptr,
			d_rigid_body_linear_momentum_ptr,
			d_rigid_body_angular_momentum_ptr,
			d_rigid_body_inertia_tensor_ptr
		);
	}
}