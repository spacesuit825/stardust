#include "hip/hip_runtime.h"
// Internal
#include "../engine.hpp"
#include "cuda_utils.hpp"
#include "collision_detection.cuh"
#include "physics_update.cuh"

// C++
#include <string>
#include <iostream>
#include <fstream>

// CUDA
#include <hip/hip_vector_types.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <nvfunctional>


namespace STARDUST {

	void DEMEngine::step(Scalar timestep) {

		// Steps
		// 1. Update sphere positions and velocities (rotation and translation) based on new particle position
		// 2. Compute relative positions of spheres to entity COM
		// 3. Compute collisions
		// 4. Compute particle force and torques from spheres
		// 5. Compute momentum (linear and angular) on particle
		// 6. Advect particle and compute quaternion

		float cell_dim = 0.2 * 2;

		int threads_per_block = 128;
		unsigned int particle_size = (m_num_particles - 1) / threads_per_block + 1;

		// PARTICLE UPDATES AND RELATIVE POSITIONS


		// COLLISION DETECTION AND RESPONSE //
		
		constructCollisionList(
			m_num_particles, 
			cell_dim, 
			d_grid_ptr, 
			d_sphere_ptr, 
			d_particle_size_ptr, 
			d_particle_position_ptr, 
			threads_per_block,
			d_temp_ptr
		);

		unsigned int count;
		hipMemcpy(&count, d_temp_ptr, sizeof(unsigned int), hipMemcpyDeviceToHost);

		std::cout << "Cells Occupied " << count << "\n";

		sortCollisionList(
			d_grid_ptr,
			d_sphere_ptr,
			d_grid_temp_ptr,
			d_sphere_temp_ptr,
			d_radices_ptr,
			d_radix_sums_ptr,
			m_num_particles
		);

		tranverseAndResolveCollisionList(
			d_grid_ptr,
			d_sphere_ptr,
			d_particle_position_ptr,
			d_particle_velocity_ptr,
			d_particle_forces_ptr,
			d_particle_mass_ptr,
			d_particle_size_ptr,
			d_particle_to_rigid_idx_ptr,
			m_num_particles,
			d_temp_ptr,
			threads_per_block
		);

		float4 force0;
		float4 force1;
		CUDA_ERR_CHECK(hipMemcpy(&force0, d_particle_forces_ptr, sizeof(float4), hipMemcpyDeviceToHost));
		CUDA_ERR_CHECK(hipMemcpy(&force1, d_particle_forces_ptr + 1, sizeof(float4), hipMemcpyDeviceToHost));

		printf("Force on Particle 0: %.3f, %.3f, %.3f\n", force0.x, force0.y, force0.z);
		printf("Force on Particle 1: %.3f, %.3f, %.3f\n", force1.x, force1.y, force1.z);

		// PARTICLE FORCE COMPUTATION AND POSITION/ORIENTATION UPDATE

	}
}