// Internal
#include "../../cuda_utils.hpp"
#include "sap_collision.cuh"

// C++
#include <string>
#include <iostream>
#include <fstream>

// CUDA
#include <hip/hip_vector_types.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <nvfunctional>

namespace STARDUST {

	void SAPCollision::reactCollisions() {

	}

}