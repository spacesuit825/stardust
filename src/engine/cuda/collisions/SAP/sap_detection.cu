#include "hip/hip_runtime.h"
// Internal
#include "../../cuda_utils.hpp"
#include "sap_collision.cuh"
#include "../../collision_detection.cuh"

// C++
#include <string>
#include <iostream>
#include <fstream>

// CUDA
#include <hip/hip_vector_types.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <nvfunctional>

namespace STARDUST {

	void SAPCollision::initiatePointers() {

		d_idx_ptr = nullptr;

		d_lower_bound_ptr = nullptr;
		d_upper_bound_ptr = nullptr;

		d_lower_extent_x_ptr = nullptr;
		d_upper_extent_x_ptr = nullptr;

		d_lower_extent_y_ptr = nullptr;
		d_upper_extent_y_ptr = nullptr;

		d_lower_extent_z_ptr = nullptr;
		d_upper_extent_z_ptr = nullptr;

		d_potential_collision_ptr = nullptr;

		d_temp_key_ptr = nullptr;
		d_temp_value_ptr = nullptr;

		d_radix_ptr = nullptr;
		d_radix_sum_ptr = nullptr;
	}

	void SAPCollision::prepareData() {

	}

	void SAPCollision::allocateCUDA(int n_spheres, int max_collisions) {

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_idx_ptr,
			n_spheres * sizeof(int)
		));

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_lower_bound_ptr,
			n_spheres * sizeof(float4)
		));

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_upper_bound_ptr,
			n_spheres * sizeof(float4)
		));

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_lower_extent_x_ptr,
			n_spheres * sizeof(float)
		));

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_upper_extent_x_ptr,
			n_spheres * sizeof(float)
		));

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_lower_extent_y_ptr,
			n_spheres * sizeof(float)
		));

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_upper_extent_y_ptr,
			n_spheres * sizeof(float)
		));

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_lower_extent_z_ptr,
			n_spheres * sizeof(float)
		));

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_upper_extent_z_ptr,
			n_spheres * sizeof(float)
		));

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_potential_collision_ptr,
			(max_collisions * n_spheres + max_collisions) * sizeof(int)
		));

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_temp_key_ptr,
			n_spheres * sizeof(float)
		));

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_temp_value_ptr,
			n_spheres * sizeof(int)
		));

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_radix_ptr,
			NUM_BLOCKS * NUM_RADICES * GROUPS_PER_BLOCK * sizeof(int)
		));

		CUDA_ERR_CHECK(hipMalloc(
			(void**)&d_radix_sum_ptr,
			NUM_RADICES * sizeof(int)
		));

	}

	void SAPCollision::transferDataToDevice() {

	}

	// Process Collisions //

	// Clear and initiate some tracking arrays
	__global__ void initIdxCUDA(
		int n_objects,
		int max_collisions,
		int* d_idx_ptr,
		int* d_potential_collision_ptr
	)
	{
		
		unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;

		if (tid >= n_objects) {
			return;
		}

		d_idx_ptr[tid] = tid;

		for (int i = 0; i < max_collisions; i++) {
			d_potential_collision_ptr[tid + i] = -1;
		}

	}

	void SAPCollision::initIdx(
		int n_objects,
		int max_collisions)
	{

		int threadsPerBlock = 256;
		int numBlocks = (n_objects + threadsPerBlock - 1) / threadsPerBlock;

		initIdxCUDA << <numBlocks, threadsPerBlock >> > (
			n_objects,
			max_collisions,
			d_idx_ptr,
			d_potential_collision_ptr
			);

	}




	// Compute object AABBs
	__global__ void computeAABBCUDA(
		int n_objects,
		float4* position,
		float* radius,
		float4* lower,
		float4* upper
	)
	{
		unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;

		if (tid >= n_objects) {
			return;
		}

		float4 pos = position[tid];
		float r = radius[tid];

		upper[tid] = make_float4(pos.x + r, pos.y + r, pos.z + r, 0.0f);
		lower[tid] = make_float4(pos.x - r, pos.y - r, pos.z - r, 0.0f);

	}

	void SAPCollision::computeAABB(
		float4* d_position_ptr, 
		float* d_radius_ptr,
		int n_objects) 
	{

		int threadsPerBlock = 256;
		int numBlocks = (n_objects + threadsPerBlock - 1) / threadsPerBlock;

		computeAABBCUDA << < numBlocks, threadsPerBlock >> > (
			n_objects,
			d_position_ptr,
			d_radius_ptr,
			d_lower_bound_ptr,
			d_upper_bound_ptr
			);

	}




	__global__ void projectAABBCUDA(
		int n_objects,
		float4* d_lower_bound_ptr,
		float4* d_upper_bound_ptr,
		float* d_lowerx_ptr,
		float* d_upperx_ptr,
		float* d_lowery_ptr,
		float* d_uppery_ptr,
		float* d_lowerz_ptr,
		float* d_upperz_ptr)
	{
		unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;

		if (tid >= n_objects) {
			return;
		}

		float4 lower_bound = d_lower_bound_ptr[tid];
		float4 upper_bound = d_upper_bound_ptr[tid];

		d_lowerx_ptr[tid] = lower_bound.x;

		d_lowery_ptr[tid] = lower_bound.y;
		d_lowerz_ptr[tid] = lower_bound.z;

		d_upperx_ptr[tid] = upper_bound.x;
		d_uppery_ptr[tid] = upper_bound.y;
		d_upperz_ptr[tid] = upper_bound.z;

	
	}

	void SAPCollision::projectAABB(
		int n_objects) 
	{

		int threadsPerBlock = 256;
		int numBlocks = (n_objects + threadsPerBlock - 1) / threadsPerBlock;

		projectAABBCUDA << < numBlocks, threadsPerBlock >> > (
			n_objects,
			d_lower_bound_ptr,
			d_upper_bound_ptr,
			d_lower_extent_x_ptr,
			d_upper_extent_x_ptr,
			d_lower_extent_y_ptr,
			d_upper_extent_y_ptr,
			d_lower_extent_z_ptr,
			d_upper_extent_z_ptr
			);
	}





	void SAPCollision::sortLowestExtents(
		int n_objects) 
	{
		SpatialPartition::sortCollisionList(
			(uint32_t*)d_lower_extent_x_ptr,
			(uint32_t*)d_idx_ptr,
			(uint32_t*)d_temp_key_ptr,
			(uint32_t*)d_temp_value_ptr,
			(uint32_t*)d_radix_ptr,
			(uint32_t*)d_radix_sum_ptr,
			n_objects
		);
	}


	__device__ void populateCollisions(
		int tid,
		int& collision_length,
		int* pending_collisions,
		int& idx)
	{

		bool unique_collision = true;
		for (int k = 0; k < 10; k++) { // <-- Max collisions is 10!!
			if (idx == pending_collisions[k]) {

				unique_collision = false;
			}
		}

		if (unique_collision == false) {
			return;
		}
		else {

			if (collision_length <= 10 && idx != tid) {
				pending_collisions[collision_length] = idx;
				collision_length += 1;
			}

			return;
		}
	}

	__global__ void sweepAndPruneCUDA(
		int n_objects,
		float* upperx,
		float* lowerx,
		float* uppery,
		float* lowery,
		float* upperz,
		float* lowerz,
		int* idxx,
		int* idxy,
		int* idxz,
		int* potential_collision,
		int padding)
	{

		unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;

		if (tid >= n_objects) {
			return;
		}

		int* idx;
		float* upper;
		float* lower;
		int* coll;

		int collision = 0;

		int coll_counter = 0;

		int pending_collisions[10];

		int home_idx = tid;
		int sorted_home_idx = idxx[home_idx];
		float home_upper_extent = upperx[sorted_home_idx];

		int phantom_idx;
		float phantom_lower_extent;

		int pending_collision_length = 0;

		int n_tid = tid + padding;
		if (n_tid >= n_objects) {
			n_tid = n_objects;
		}
		else {
			
		}

		for (int i = tid + 1; i < n_tid; i++) {

			if (i == home_idx) {
				continue;
			}

			phantom_lower_extent = lowerx[i];

			phantom_idx = idxx[i];

			// Check X proj
			if (phantom_lower_extent <= home_upper_extent) { // <-- TODO: change this so it starts with axis with most position variance

				home_upper_extent = uppery[sorted_home_idx];
				phantom_lower_extent = lowery[phantom_idx];

				// Check Y proj
				if (phantom_lower_extent <= home_upper_extent) {

					home_upper_extent = upperz[sorted_home_idx];
					phantom_lower_extent = lowerz[phantom_idx];

					// Check Z proj
					if (phantom_lower_extent <= home_upper_extent) {

						coll_counter += 1;
						populateCollisions(tid, pending_collision_length, potential_collision + tid * padding, phantom_idx);
						//printf("Collision detected between: %d and %d\n", sorted_home_idx, phantom_idx);

					}
				}
			}
		}
	}

	void SAPCollision::sweepAndPrune(
		int n_objects) 
	{

		int threadsPerBlock = 256;
		int numBlocks = (n_objects + threadsPerBlock - 1) / threadsPerBlock;

		sweepAndPruneCUDA << <numBlocks, threadsPerBlock >> > (
			n_objects,
			d_upper_extent_x_ptr,
			d_temp_key_ptr,
			d_upper_extent_y_ptr,
			d_lower_extent_y_ptr,
			d_upper_extent_z_ptr,
			d_lower_extent_z_ptr,
			d_temp_value_ptr,
			d_idx_ptr,
			d_idx_ptr,
			d_potential_collision_ptr,
			10);

	}

	void SAPCollision::processCollisions(
		float4* d_position_ptr,
		float* d_radius_ptr,
		int n_objects,
		int max_collisions
	) 
	{
		SAPCollision::initIdx(
			n_objects,
			max_collisions
		);

		

		SAPCollision::computeAABB(
			d_position_ptr,
			d_radius_ptr,
			n_objects
		);

		

		SAPCollision::projectAABB(
			n_objects
		);

		

		SAPCollision::sortLowestExtents(
			n_objects
		);



		SAPCollision::sweepAndPrune(
			n_objects
		);



		CUDA_ERR_CHECK(hipDeviceSynchronize());
	}

}
