// Internal
#include "../engine.hpp"
#include "cuda_utils.hpp"
#include "collision_detection.cuh"

// C++
#include <string>
#include <iostream>
#include <fstream>

// CUDA
#include <hip/hip_vector_types.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <nvfunctional>


namespace STARDUST {

	// Template for launching kernels
	template<typename... Arguments>
	void KernelLaunch(std::string&& tag, int gs, int bs, void(*f)(Arguments...), Arguments... args) {
		f << <gs, bs >> > (args...);

		CUDA_ERR_CHECK(hipPeekAtLastError());
		CUDA_ERR_CHECK(hipDeviceSynchronize());
	}





	void DEMEngine::step(Scalar timestep) {

		float cell_dim = 0.5;

		int threads_per_block = 128;
		unsigned int particle_size = (m_num_particles - 1) / threads_per_block + 1;
		
		constructCells(
			m_num_particles, 
			cell_dim, 
			d_grid_ptr, 
			d_sphere_ptr, 
			d_particle_size_ptr, 
			d_particle_position_ptr, 
			threads_per_block,
			d_temp_ptr);

		unsigned int count;
		hipMemcpy(&count, d_temp_ptr, sizeof(unsigned int), hipMemcpyDeviceToHost);

		std::cout << count << "\n";


	}
}