#include "hip/hip_runtime.h"
// C++
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <stdint.h>

// Thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/sort.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

// CUDA
#include <hip/hip_vector_types.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <nvfunctional>
#include <hip/hip_runtime_api.h>

#include <iostream>

void debugVector(thrust::host_vector<float4>& vector) {
    std::cout << "\nDebug Vector Print\n" << "----------------------------\n";
    for (int i = 0; i < vector.size(); i++) {
        std::cout << "Vector " << i << ": ";
        std::cout << "[" << vector[i].x << ", " << vector[i].y << ", " << vector[i].z << "]\n";
    }
}

void debugList(std::string name, thrust::device_vector<float>& vector) {
    std::cout << "\n" << name << "\n" << "----------------------------\n";
    std::cout << "[";
    for (int i = 0; i < vector.size(); i++) {
        std::cout << vector[i] << " ";
    }
    std::cout << "]\n";
}

void debugList(std::string name, thrust::device_vector<int>& vector) {
    std::cout << "\n" << name << "\n" << "----------------------------\n";
    std::cout << "[";
    for (int i = 0; i < vector.size(); i++) {
        std::cout << vector[i] << " ";
    }
    std::cout << "]\n";
}

void computeAABB(
    thrust::host_vector<float4>& position, 
    thrust::host_vector<float>& radius, 
    thrust::host_vector<float4>& lower_bound, 
    thrust::host_vector<float4>& upper_bound) 
{
    for (int i = 0; i < position.size(); i++) {
        float4 pos = position[i];
        float r = radius[i];

        upper_bound[i] = make_float4(pos.x + r, pos.y + r, pos.z + r, 0.0f);
        lower_bound[i] = make_float4(pos.x - r, pos.y - r, pos.z - r, 0.0f);
    }
}

struct project_functorx
{
    __host__ __device__
        float operator()(const float4& x) const
    {
        return x.x;
    }
};

struct project_functory
{
    __host__ __device__
        float operator()(const float4& x) const
    {
        return x.x;
    }
};

struct project_functorz
{
    __host__ __device__
        float operator()(const float4& x) const
    {
        return x.x;
    }
};

void projectAABBx(
    thrust::device_vector<float4>& lower_bound,
    thrust::device_vector<float4>& upper_bound,
    thrust::device_vector<float>& lower,
    thrust::device_vector<float>& upper)
{
    thrust::transform(lower_bound.begin(), lower_bound.end(), lower.begin(), project_functorx());
    thrust::transform(upper_bound.begin(), upper_bound.end(), upper.begin(), project_functorx());
}

void projectAABBy(
    thrust::device_vector<float4>& lower_bound,
    thrust::device_vector<float4>& upper_bound,
    thrust::device_vector<float>& lower,
    thrust::device_vector<float>& upper)
{
    thrust::transform(lower_bound.begin(), lower_bound.end(), lower.begin(), project_functory());
    thrust::transform(upper_bound.begin(), upper_bound.end(), upper.begin(), project_functory());
}

void projectAABBz(
    thrust::device_vector<float4>& lower_bound,
    thrust::device_vector<float4>& upper_bound,
    thrust::device_vector<float>& lower,
    thrust::device_vector<float>& upper)
{
    thrust::transform(lower_bound.begin(), lower_bound.end(), lower.begin(), project_functorz());
    thrust::transform(upper_bound.begin(), upper_bound.end(), upper.begin(), project_functorz());
}

void radixSort(
    thrust::device_vector<float>& lower,
    thrust::device_vector<int>& idx)
{
    thrust::sort_by_key(lower.begin(), lower.end(), idx.begin());
}

__device__ void populateCollisions(
    int tid, 
    int& collision_length, 
    int* pending_collisions, 
    int& idx)
{

    bool unique_collision = true;
    for (int k = 0; k < 10; k++) { // <-- Max collisions is 10!!
        if (idx == pending_collisions[k]) {

            unique_collision = false;
        }
    }

    if (unique_collision == false) {
        return;
    }
    else {
        
        pending_collisions[collision_length + 1] = idx;
        collision_length += 1;

        return;
    }
}

__global__ void sweepBlocks(
    float* upperx,
    float* lowerx,
    float* uppery,
    float* lowery,
    float* upperz,
    float* lowerz,
    int* idxx,
    int* idxy,
    int* idxz,
    int* potential_collision,
    int n_objects,
    int padding)
{
    // Shared memory
    extern __shared__ int collisions[];

    int obj_idx = blockIdx.x;
    int sorted_home_idx = idxx[obj_idx];
    float home_upper_extent = upperx[sorted_home_idx];

    int phantom_idx;
    float phantom_lower_extent;

    int phantom_position = threadIdx.x + blockIdx.x;

    if (phantom_position >= n_objects) {
        return;
    }

    if (phantom_position == sorted_home_idx) {
        return;
    }

    phantom_idx = idxx[phantom_position];

    phantom_lower_extent = lowerx[phantom_position];

    

    // Check X proj
    if (phantom_lower_extent <= home_upper_extent) { // <-- TODO: change this so it starts with axis with most position variance

        home_upper_extent = uppery[sorted_home_idx];
        phantom_lower_extent = lowery[phantom_idx];

        // Check Y proj
        if (phantom_lower_extent <= home_upper_extent) {

            home_upper_extent = upperz[sorted_home_idx];
            phantom_lower_extent = lowerz[phantom_idx];

            // Check Z proj
            if (phantom_lower_extent <= home_upper_extent) {

                printf("collision detected between %d and %d\n", sorted_home_idx, phantom_idx);

            }
        }
    }


}

__global__ void sweep(
    float* upperx,
    float* lowerx,
    float* uppery,
    float* lowery,
    float* upperz,
    float* lowerz,
    int* idxx,
    int* idxy,
    int* idxz,
    int* potential_collision,
    int n_objects,
    int padding)
{

    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid >= n_objects) {
        return;
    }

    int* idx;
    float* upper;
    float* lower;
    int* coll;

    int collision = 0;

    int pending_collisions[10];

    int home_idx = tid;
    int sorted_home_idx = idxx[home_idx];
    float home_upper_extent = upperx[sorted_home_idx];

    int phantom_idx;
    float phantom_lower_extent;

    int pending_collision_length = 0;

    int n_tid = tid + 10;
    if (n_tid > n_objects) {
        n_tid = n_objects;
    }
    else {
        n_tid = 10;
    }

    for (int i = tid + 1; i < n_tid; i++) {

        if (i == sorted_home_idx) {
            continue;
        }

        phantom_lower_extent = lowerx[i];

        phantom_idx = idxx[i];

        // Check X proj
        if (phantom_lower_extent <= home_upper_extent) { // <-- TODO: change this so it starts with axis with most position variance

            home_upper_extent = uppery[sorted_home_idx];
            phantom_lower_extent = lowery[phantom_idx];

            // Check Y proj
            if (phantom_lower_extent <= home_upper_extent) {

                home_upper_extent = upperz[sorted_home_idx];
                phantom_lower_extent = lowerz[phantom_idx];

                // Check Z proj
                if (phantom_lower_extent <= home_upper_extent) {

                    populateCollisions(tid, pending_collision_length, potential_collision + tid * padding, phantom_idx);
                    //printf("Collision detected between: %d and %d\n", sorted_home_idx, phantom_idx);

                }
            }
        }
    }

}

struct isValid
{
    __host__ __device__
        bool operator()(const int& value) const
    {
        return value >= 0;
    }
};

float randFloat(float a, float b) {
    float random = ((float)rand()) / (float)RAND_MAX;
    float diff = b - a;
    float r = random * diff;
    return a + r;
}

int main() {

    int n_objects = 2;
    int max_collisions = 10;

     //H has storage for 4 integers
    thrust::host_vector<float4> position(n_objects);
    thrust::host_vector<float> radius(n_objects);
    thrust::host_vector<float4> lower_bound(n_objects);
    thrust::host_vector<float4> upper_bound(n_objects);

    thrust::host_vector<float> lowerx(n_objects);
    thrust::host_vector<float> upperx(n_objects);
    thrust::host_vector<float> lowery(n_objects);
    thrust::host_vector<float> uppery(n_objects);
    thrust::host_vector<float> lowerz(n_objects);
    thrust::host_vector<float> upperz(n_objects);

    thrust::host_vector<int> idxx(n_objects);
    thrust::host_vector<int> idxy(n_objects);
    thrust::host_vector<int> idxz(n_objects);

    thrust::host_vector<int> potential_collision(max_collisions * n_objects + max_collisions);

    /*for (int i = 0; i < n_objects; i++) {
        position[i] = make_float4(randFloat(0.5, 10.0), randFloat(0.5, 10.0), randFloat(0.5, 10.0), 0.0f);
        radius[i] = randFloat(0.4, 0.5);
        idxx[i] = i;
        idxy[i] = i;
        idxz[i] = i;
        
    }*/

    position[0] = make_float4(0.0, 0.0, 0.0, 0.0);
    position[1] = make_float4(0.0, 0.0, 0.5, 0.0);
    //position[2] = make_float4(0.0, 0.0, 0.5, 0.0);
    radius[0] = 0.5;
    radius[1] = 0.5;
   // radius[2] = 0.5;
    idxx[0] = 0;
    idxy[0] = 0;
    idxz[0] = 0;
    idxx[1] = 1;
    idxy[1] = 1;
    idxz[1] = 1;
    //idxx[2] = 2;
    //idxy[2] = 2;
    //idxz[2] = 2;

    for (int i = 0; i < max_collisions * n_objects + max_collisions; i++) {
        potential_collision[i] = -1;
    }

    computeAABB(position, radius, lower_bound, upper_bound);

    //// Copy host_vector H to device_vector D
    thrust::device_vector<float4> d_position = position;
    thrust::device_vector<float> d_radius = radius;
    thrust::device_vector<float4> d_lower_bound = lower_bound;
    thrust::device_vector<float4> d_upper_bound = upper_bound;

    thrust::device_vector<float> d_lowerx = lowerx;
    thrust::device_vector<float> d_upperx = upperx;
    thrust::device_vector<float> d_lowery = lowery;
    thrust::device_vector<float> d_uppery = uppery;
    thrust::device_vector<float> d_lowerz = lowerz;
    thrust::device_vector<float> d_upperz = upperz;

    thrust::device_vector<int> d_idxx = idxx;
    thrust::device_vector<int> d_idxy = idxy;
    thrust::device_vector<int> d_idxz = idxz;

    thrust::device_vector<int> d_potential_collision = potential_collision;

    // First cast all device_vectors to pointers
    float4* d_position_ptr = thrust::raw_pointer_cast(d_position.data());
    float4* d_lower_bound_ptr = thrust::raw_pointer_cast(d_lower_bound.data());
    float4* d_upper_bound_ptr = thrust::raw_pointer_cast(d_upper_bound.data());
    float* d_radius_ptr = thrust::raw_pointer_cast(d_radius.data());

    float* d_upperx_ptr = thrust::raw_pointer_cast(d_upperx.data());
    float* d_lowerx_ptr = thrust::raw_pointer_cast(d_lowerx.data());
    float* d_uppery_ptr = thrust::raw_pointer_cast(d_uppery.data());
    float* d_lowery_ptr = thrust::raw_pointer_cast(d_lowery.data());
    float* d_upperz_ptr = thrust::raw_pointer_cast(d_upperz.data());
    float* d_lowerz_ptr = thrust::raw_pointer_cast(d_lowerz.data());

    int* d_idxx_ptr = thrust::raw_pointer_cast(d_idxx.data());
    int* d_idxy_ptr = thrust::raw_pointer_cast(d_idxy.data());
    int* d_idxz_ptr = thrust::raw_pointer_cast(d_idxz.data());

    int* d_potential_collision_ptr = thrust::raw_pointer_cast(d_potential_collision.data());

    int threadsPerBlock = 256;
    int numBlocks = (n_objects + threadsPerBlock - 1) / threadsPerBlock;

    std::chrono::time_point<std::chrono::system_clock> start;
    std::chrono::duration<double> duration;

    double time;
    start = std::chrono::system_clock::now();

    

    

    // Project to x-axis
    projectAABBx(d_lower_bound, d_upper_bound, d_lowerx, d_upperx);
    projectAABBy(d_lower_bound, d_upper_bound, d_lowery, d_uppery);
    projectAABBz(d_lower_bound, d_upper_bound, d_lowerz, d_upperz);

    // Radix sort
    radixSort(d_lowerx, d_idxx);
    //radixSort(d_lowery, d_idxy);
    //radixSort(d_lowerz, d_idxz);

    // Perform the sweep
   /* int threadsPerBlock = 256;
    int numBlocks = (n_objects + threadsPerBlock - 1) / threadsPerBlock;*/

    //std::cout << "Launching kernel\n";
    /*sweep << <numBlocks, threadsPerBlock >> > (
        d_upperx_ptr,
        d_lowerx_ptr,
        d_upperx_ptr,
        d_lowerx_ptr,
        d_upperx_ptr,
        d_lowerx_ptr,
        d_idxx_ptr,
        d_idxy_ptr,
        d_idxz_ptr,
        d_potential_collision_ptr,
        n_objects,
        max_collisions);*/

    sweepBlocks << <n_objects, threadsPerBlock >> > (
        d_upperx_ptr,
        d_lowerx_ptr,
        d_upperx_ptr,
        d_lowerx_ptr,
        d_upperx_ptr,
        d_lowerx_ptr,
        d_idxx_ptr,
        d_idxy_ptr,
        d_idxz_ptr,
        d_potential_collision_ptr,
        n_objects,
        max_collisions);

    //std::cout << "Kernel completed\n";

    duration = std::chrono::system_clock::now() - start;

    time = duration.count();

    

    int sum = thrust::count_if(d_potential_collision.begin(), d_potential_collision.end(), isValid());

    std::cout << sum << " Collisions detected in " << time << " secs\n" << std::endl;
    // int sum = thrust::reduce(d_potential_collision.begin(), d_potential_collision.end(), 0, thrust::plus<int>());

    std::cout << "Number of objects: " << n_objects << "\n";
    



    return 0;
}